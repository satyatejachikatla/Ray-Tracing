#include "hip/hip_runtime.h"
#include <Render.h>


__device__ vec3 color(const ray& r) {
	vec3 unit_direction = unit_vector(r.direction());
	float t = 0.5f*(unit_direction.y() + 1.0f);
	return (1.0f-t)*vec3(1.0f, 1.0f, 1.0f) + t*vec3(0.5f, 0.7f, 1.0f);
}

__global__ void render(vec3 *fb, int max_x, int max_y, vec3 lower_left_corner, vec3 horizontal, vec3 vertical, vec3 origin) {

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if((i >= max_x) || (j >= max_y)) return;

	int pixel_index = j*max_x + i;
	float u = float(i) / float(max_x);
	float v = float(j) / float(max_y);
	ray r(origin, lower_left_corner + u*horizontal + v*vertical);
	fb[pixel_index] = color(r);
}