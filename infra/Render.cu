#include "hip/hip_runtime.h"
#include <cfloat>

#include <Render.h>
#include <Hitable.h>
#include <Material.h>
#include <Objects/Sphere.h>

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j*max_x + i;
	//Each thread gets same seed, a different sequence number, no offset
	hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void create_world(hitable **d_list, hitable **d_world, camera **d_camera) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		*(d_list)   = new sphere(vec3(0,0,-1), 0.5,
									new lambertian(vec3(0.8,0.3,0.3)));
		*(d_list+1) = new sphere(vec3(0,-100.5,-1), 100,
									new lambertian(vec3(0.8,0.8,0.0)));
		*d_world    = new hitable_list(d_list,2);
		*d_camera   = new camera();
	}
}

__global__ void free_world(hitable **d_list, hitable **d_world, camera **d_camera) {
	delete ((sphere*)d_list[0])->mat_ptr;
	delete ((sphere*)d_list[1])->mat_ptr;
	delete  d_list[0];
	delete  d_list[1];
	delete *d_world;
	delete *d_camera;
}

__device__ vec3 color(const ray& r, hitable **world,hiprandState *local_rand_state) {
	ray cur_ray = r;
	vec3 cur_attenuation(1.0f,1.0f,1.0f);
	for(int i = 0; i < 50; i++) { // Here 50 bounces of ray is max
		hit_record rec;
		if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
			ray scattered;
			vec3 attenuation;
			if(rec.mat_ptr->scatter(cur_ray,rec,attenuation,scattered,local_rand_state)){
				cur_attenuation *= attenuation;
				cur_ray = scattered;
			}
			else {
				return vec3(0.0f,0.0f,0.0f);
			}
		}
		else {
			vec3 unit_direction = unit_vector(cur_ray.direction());
			float t = 0.5f*(unit_direction.y() + 1.0f);
			vec3 c = (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
			return cur_attenuation * c;
		}
	}
	return vec3(0.0,0.0,0.0); // exceeded recursion
}

__global__ void render(vec3 *fb, int max_x, int max_y, int ns, camera **cam, hitable **world, hiprandState *rand_state) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if((i >= max_x) || (j >= max_y)) return;

	int pixel_index = j*max_x + i;
	hiprandState local_rand_state = rand_state[pixel_index];
	vec3 col(0,0,0);
	for(int s=0; s < ns; s++) {
		float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
		float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
		ray r = (*cam)->get_ray(u,v);
		col += color(r, world,&local_rand_state);
	}
	fb[pixel_index] = col/float(ns);
}

