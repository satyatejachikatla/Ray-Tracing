#include "hip/hip_runtime.h"
#include <cfloat>

#include <Render.h>
#include <Hitable.h>
#include <Objects/Sphere.h>


__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if((i >= max_x) || (j >= max_y)) return;
	int pixel_index = j*max_x + i;
	//Each thread gets same seed, a different sequence number, no offset
	hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void create_world(hitable **d_list, hitable **d_world, camera **d_camera) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		*(d_list)   = new sphere(vec3(0,0,-1), 0.5);
		*(d_list+1) = new sphere(vec3(0,-100.5,-1), 100);
		*d_world    = new hitable_list(d_list,2);
		*d_camera   = new camera();
	}
}

__global__ void free_world(hitable **d_list, hitable **d_world, camera **d_camera) {
	delete *(d_list);
	delete *(d_list+1);
	delete *d_world;
	delete *d_camera;
}




#define RANDVEC3 vec3(hiprand_uniform(local_rand_state),hiprand_uniform(local_rand_state),hiprand_uniform(local_rand_state))

__device__ vec3 random_in_unit_sphere(hiprandState *local_rand_state) {
	vec3 p;
	do {
		p = 2.0f*RANDVEC3 - vec3(1,1,1);
	} while (p.squared_length() >= 1.0f);
	return p;
}



__device__ vec3 color(const ray& r, hitable **world,hiprandState *local_rand_state) {
	ray cur_ray = r;
	float cur_attenuation = 1.0f;
	for(int i = 0; i < 50; i++) { // Here 50 bounces of ray is max
		hit_record rec;
		if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
			vec3 target = rec.p + rec.normal + random_in_unit_sphere(local_rand_state);
			cur_attenuation *= 0.5f;
			cur_ray = ray(rec.p, target-rec.p);
		}
		else {
			vec3 unit_direction = unit_vector(cur_ray.direction());
			float t = 0.5f*(unit_direction.y() + 1.0f);
			vec3 c = (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
			return cur_attenuation * c;
		}
	}
	return vec3(0.0,0.0,0.0); // exceeded recursion
}

__global__ void render(vec3 *fb, int max_x, int max_y, int ns, camera **cam, hitable **world, hiprandState *rand_state) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if((i >= max_x) || (j >= max_y)) return;

	int pixel_index = j*max_x + i;
	hiprandState local_rand_state = rand_state[pixel_index];
	vec3 col(0,0,0);
	for(int s=0; s < ns; s++) {
		float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
		float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
		ray r = (*cam)->get_ray(u,v);
		col += color(r, world,&local_rand_state);
	}
	fb[pixel_index] = col/float(ns);
}

