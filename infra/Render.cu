#include "hip/hip_runtime.h"
#include <Render.h>
#include <Vector.h>
#include <Ray.h>

__device__ vec3 color(const ray& r) {
   vec3 unit_direction = unit_vector(r.direction());
   float t = 0.5f*(unit_direction.y() + 1.0f);
   return (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
}

__global__ void render(float *fb, int max_x, int max_y) {

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if((i >= max_x) || (j >= max_y)) return;
	// 3 is because of rgb //
	int pixel_index = j*max_x*3 + i*3;
	fb[pixel_index + 0] = float(i) / max_x;
	fb[pixel_index + 1] = float(j) / max_y;
	fb[pixel_index + 2] = 0.2;
}